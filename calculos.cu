#include "hip/hip_runtime.h"
/*

 Universidad del Valle de Guatemala
 CC3056
 Ana Lucia Hernandez. 17138.
 Andrea Arg�ello. 17801.
 Proyecto 3
 Programaci�n de Microprocesadores

*/


#include <iostream>
#include <stdio.h>
#include <string>
#include <string.h>
#include <errno.h>
#include <stdint.h>
#include <stdlib.h>
#include <time.h>
#include <io.h>
#include <hip/hip_runtime.h>
#include <fstream>

#define N 192 //cantidad de datos en los dos dias
#define T 96 //cantidad de datos del tercer dia

using namespace std;

/* El vector y guarda 96 predicciones
 * El vector x contiene los numeros de 0-95, o sea el numero de medicion de 15 minutos a evaluar
 * x192 contiene las mediciones de 15 minutos de los primeros dos dias
 * y192 contiene el valor en y (temperatura, humedad o presion) de los primeros dos dias
 */
__global__ void prediccion(float *y, float *x, float *x192, float *y192)
{
    int myID= (int)threadIdx.x; //
    float m=(y192[myID]-y192[myID+T])/(x192[myID]-x192[myID+T]); //pendiente
    float b=(m*x192[myID])+y192[myID]; //intercepto
    y[myID] = ((float)m*x[myID]) + (float)b; //ecuacion de una recta
}

__global__ void porcentajeError(float *resultado, float *teorico, float *predic)//los tres arrays son/seran de 96 datos
{
    int myID = (int)threadIdx.x; //deberian ser 96
    resultado[myID]=(float) (predic[myID]-teorico[myID])*100/teorico[myID];
}

int main(int argv, char* argc[])
{
    /* creacion streams, un stream por variable
     * (temperatura, presion, humedad) y sus respectivos % de error */
    hipStream_t stream1, stream2, stream3, stream4, stream5, stream6;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
	  hipStreamCreate(&stream4);
    hipStreamCreate(&stream5);
    hipStreamCreate(&stream6);


    /* variables del host */
    float *dev_temp, *dev_hum, *dev_pres, *dev_secs, *dev_secs3, *dev_errorTemp, *dev_errorHum, *dev_errorPres; // pointers del device
    float *dev_temp3, *dev_hum3, *dev_pres3, *dev_tempres, *dev_humres, *dev_presres, *dev_phum, *dev_ppres, *dev_ptemp;
    float *temp, *hum, *pres, *secs, *temp_res, *hum_res, *pres_res, *temp3, *hum3, *pres3;
    string fechas[N], fechas3[T];
    float *errorTemp,*errorHum,*errorPres,secs3[T];

    /* reservas en memoria de los arrays a utilizar en host y pasar al device */
    hipHostAlloc( (void**)&secs, N * sizeof(int), hipHostMallocDefault );// reserva de memoria de segundos
    hipHostAlloc( (void**)&secs3, T * sizeof(int), hipHostMallocDefault );//segundos del tercer dia
    hipHostAlloc( (void**)&pres3, T * sizeof(int), hipHostMallocDefault );//presion teorica dia 3
    hipHostAlloc( (void**)&temp3, T * sizeof(int), hipHostMallocDefault );//temperatura teorica dia 3
    hipHostAlloc( (void**)&hum3, T * sizeof(int), hipHostMallocDefault);//humedad teorica dia 3

    /*reservas en memoria de arrays de segundos y resultados del device*/
    hipMalloc( (void**)&dev_secs, N * sizeof(int) );
    hipMalloc( (void**)&dev_secs3, T * sizeof(int) );
    hipMalloc( (void**)&dev_humres, T * sizeof(int) );//prediccion de humedad
    hipMalloc( (void**)&dev_tempres, T * sizeof(int) );//prediccion de temperatura
    hipMalloc( (void**)&dev_presres, T * sizeof(int) );//prediccion de presion


    //stream 1
    hipMalloc( (void**)&dev_temp, N * sizeof(int) );
    hipHostAlloc( (void**)&temp, N * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&temp_res, T * sizeof(int), hipHostMallocDefault );



    //stream 2
    hipMalloc( (void**)&dev_hum, N * sizeof(int) );
    hipHostAlloc( (void**)&hum, N * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&hum_res, T * sizeof(int), hipHostMallocDefault );


    //stream 3
    hipMalloc( (void**)&dev_pres, N * sizeof(int) );
    hipHostAlloc( (void**)&pres_res, T * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&pres, N * sizeof(int), hipHostMallocDefault);

    //stream 4
    hipMalloc( (void**)&dev_hum3, T * sizeof(int) ); //humedad dia 3
    hipMalloc( (void**)&dev_errorHum, T * sizeof(int) ); // % error device
    hipHostAlloc( (void**)&errorHum, T * sizeof(int), hipHostMallocDefault ); // % error
    hipMalloc( (void**)&dev_phum, T * sizeof(int) ); //prediccion de humedad en device

    //stream 5
    hipMalloc( (void**)&dev_ppres, T * sizeof(int) ); //prediccion de presion en device
    hipMalloc( (void**)&dev_pres3, T * sizeof(int) ); //presion dia 3
    hipMalloc( (void**)&dev_errorPres, T * sizeof(int) ); // % error device
    hipHostAlloc( (void**)&errorPres, T * sizeof(int), hipHostMallocDefault ); // % error


    //stream 6
    hipMalloc( (void**)&dev_ptemp, T * sizeof(int) ); //prediccion de temperatura en device
    hipMalloc( (void**)&dev_temp3, T * sizeof(int) ); //temperatura dia 3
    hipMalloc( (void**)&dev_errorTemp, T * sizeof(int) ); // % error device
    hipHostAlloc( (void**)&errorTemp, T * sizeof(int), hipHostMallocDefault ); // % error


    /* lectura de datos del csv dias 1 y 2*/
    int i = 0; //indice
    string humedad, presion, temperatura, altitud, fecha;
    ifstream file("datos.csv");
    while (getline(file, humedad, ',')) {
        hum[i] = (float)atof(humedad.c_str());
        getline(file, presion, ',') ;
        pres[i] = (float)atof(presion.c_str());
        getline(file, temperatura, ',') ;
        temp[i] = (float)atof(temperatura.c_str());
        getline(file, altitud, ','); //no se almacena
        getline(file, fecha);
        fechas[i] = fecha;
        i++;
    }


    /* lectura de datos del csv del tercer dia*/
    int j = 0; //indice
    ifstream file3("dia3.csv");
    while (getline(file3, humedad, ',')) {
        hum3[j] = (float)atof(humedad.c_str());
        getline(file3, presion, ',') ;
        pres3[j] = (float)atof(presion.c_str());
        getline(file3, temperatura, ',') ;
        temp3[j] = (float)atof(temperatura.c_str());
        getline(file3, altitud, ',');
        getline(file3, fecha);
        fechas3[j] = fecha;
        j++;
    }


    /* segundos de las primeras 48hr */
    for (int i =0; i <N; i++)
    {
        secs[i]=i;
    }


    /* ajustar el vector de segundos para que ahora sean los segundos del tercer dia (prediccion) */
    for (int i =0; i <T; i++)
    {
        secs3[i] = (i+N); //48hr + segs del tercer dia
    }

    /* lanzamiento de kernels para la prediccion
     * se lanzaran N threads en los que cada uno calculara
     * la prediccion en la hora correspondiente de su variable correspondiente. */
    hipMemcpyAsync(dev_hum,hum,N*sizeof(int),hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(dev_pres,pres,N*sizeof(int),hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(dev_temp,temp,N*sizeof(int),hipMemcpyHostToDevice,stream3);
    hipMemcpyAsync(dev_secs,secs,N*sizeof(int),hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(dev_secs,secs,N*sizeof(int),hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(dev_secs,secs,N*sizeof(int),hipMemcpyHostToDevice,stream3);
    hipMemcpyAsync(dev_secs3,secs3,T*sizeof(int),hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(dev_secs3,secs3,T*sizeof(int),hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(dev_secs3,secs3,T*sizeof(int),hipMemcpyHostToDevice,stream3);

    prediccion<<<1, T, 0, stream1>>>(dev_humres, dev_secs3, dev_secs, dev_hum);
    prediccion<<<1, T, 1, stream2>>>(dev_presres, dev_secs3, dev_secs, dev_pres);
    prediccion<<<1, T, 2, stream3>>>(dev_tempres, dev_secs3, dev_secs, dev_temp);
	hipMemcpyAsync(hum_res,dev_humres,T*sizeof(int),hipMemcpyDeviceToHost,stream1);
    hipMemcpyAsync(pres_res,dev_presres,T*sizeof(int),hipMemcpyDeviceToHost,stream2);
    hipMemcpyAsync(temp_res,dev_tempres,T*sizeof(int),hipMemcpyDeviceToHost,stream3);


    hipStreamSynchronize(stream1); // wait for stream1 to finish
    hipStreamSynchronize(stream2); // wait for stream2 to finish
    hipStreamSynchronize(stream3); // wait for stream3 to finish

    /* realizacion y lanzamiento de kernels de porcentaje de error */
    hipMemcpyAsync(dev_phum,hum_res,T*sizeof(int),hipMemcpyHostToDevice,stream4);
    hipMemcpyAsync(dev_ppres,pres_res,T*sizeof(int),hipMemcpyHostToDevice,stream5);
    hipMemcpyAsync(dev_ptemp,temp_res,T*sizeof(int),hipMemcpyHostToDevice,stream6);
    hipMemcpyAsync(dev_hum3,hum3,T*sizeof(int),hipMemcpyHostToDevice,stream4);
    hipMemcpyAsync(dev_pres3,pres3,T*sizeof(int),hipMemcpyHostToDevice,stream5);
    hipMemcpyAsync(dev_temp3,temp3,T*sizeof(int),hipMemcpyHostToDevice,stream6);

    //3 kernels de un bloque de T hilos, un hilo por cada dato

	  porcentajeError<<<1, T, 0, stream4>>>(dev_errorHum, dev_hum3, dev_phum);
	  porcentajeError<<<1, T, 0, stream5>>>(dev_errorPres, dev_pres3, dev_ppres);
    porcentajeError<<<1, T, 0, stream6>>>(dev_errorTemp, dev_temp3, dev_ptemp);

	  hipMemcpyAsync(errorHum,dev_errorHum,T*sizeof(int),hipMemcpyDeviceToHost,stream4);
    hipMemcpyAsync(errorPres,dev_errorPres,T*sizeof(int),hipMemcpyDeviceToHost,stream5);
    hipMemcpyAsync(errorTemp,dev_errorTemp,T*sizeof(int),hipMemcpyDeviceToHost,stream6);

    //Esperar a finalizacion de streams
    hipStreamSynchronize(stream5);
    hipStreamSynchronize(stream4);
    hipStreamSynchronize(stream6);


    //Impresion de datos
    printf("\t\t\t\t\t\t\tPROYECTO FINAL MICROPROCESADORES");
    printf("\n\t\t\t\t\t\tAndrea Arguello 17801 \t Ana Lucia Hernandez 17138\n");

    printf("\n|\t\t\t\t\t\t\tDATOS TOMADOS DE LOS PRIMEROS DOS DIAS\t\t\t\t\t\t|");
    printf("\n|    Minutos\t|\t\t\t\tDIA 1\t\t\t|\t\t\t\tDIA 2\t\t\t|");
    for(int i=0; i<T; i++){
      printf("\n|\t%d.\t|\tH: %.2f\tP: %.2f\tT: %.2f\t|\tH: %.2f\tP: %.2f\tT: %.2f\t|",i*15,hum[i],pres[i],temp[i],hum[i+T],pres[i+T],temp[i+T]);
    }

    printf("\n\n\n|\t\t\t\t\t\t\t\t\t\tPREDICCIONES Y VALORES TEORICOS DEL TERCER DIA\t\t\t\t\t\t\t\t\t\t|");
    printf("\n|Min.\t|\t\t\t\tHUMEDAD\t\t\t\t|\t\t\t\tPRESION\t\t\t\t|\t\t\t\tTEMPERATURA\t\t\t|\n");
    ofstream MiArchivo ("prediccion.csv");
    for(int i=0; i<T; i++)
    {
        if (MiArchivo.is_open())
        {
            MiArchivo <<hum_res[i]<<","<< errorHum[i]<<","<<pres_res[i]<<","<<errorPres[i]<<","<<temp_res[i]<<","<<errorTemp[i]<<","<<(secs[i]+192)*900<<"\n";
            printf("| %d.\t|  Teorica %.2f\tPrediccion %.2f (%.2f%% de error)\t|\tTeorica %.2f\tPrediccion %.2f (%.2f%% de error)\t|\tTeorica %.2f\tExperimental %.2f (%.2f%% de error)\t|\n",(int)secs[i]*15, hum3[i], hum_res[i], errorHum[i], pres3[i], pres_res[i], errorPres[i], temp3[i], temp_res[i], errorTemp[i]);
        }
    }
    MiArchivo.close();

    printf("TIEMPOS DE EJECUCION");

    //Destruccion de streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipStreamDestroy(stream4);
    hipStreamDestroy(stream5);
    hipStreamDestroy(stream6);

    //Liberar memoria
    hipFree(dev_hum);
    hipFree(dev_temp);
    hipFree(dev_pres);
    hipFree(dev_secs);
    hipFree(dev_humres);
    hipFree(dev_tempres);
    hipFree(dev_presres);
    hipFree(dev_secs3);
    hipFree(dev_ppres);
    hipFree(dev_ptemp);
    hipFree(dev_phum);
    hipFree(dev_pres3);
    hipFree(dev_temp3);
    hipFree(dev_hum3);
    hipFree(dev_errorTemp);
    hipFree(dev_errorPres);
    hipFree(dev_errorHum);

    //Fin
    return 0;
}
