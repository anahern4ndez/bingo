#include "hip/hip_runtime.h"
/*
 
 Universidad del Valle de Guatemala
 CC3056
 Ana Lucia Hernandez. 17138.
 Andrea Arg�ello. 17801.
 Proyecto 3
 Programaci�n de Microprocesadores
 
*/


#include <iostream>
#include <stdio.h>
#include <string>
#include <string.h>
#include <errno.h>
#include <stdint.h>
#include <stdlib.h>
#include <time.h>
#include <io.h>
#include <hip/hip_runtime.h>
#include <fstream>

#define N 192 //cantidad de datos en los dos dias (serian 192)
#define T 96 //cantidad de datos del tercer dia (serian 96)
using namespace std;

struct regresionLineal
{
    double pendiente;
    double intercepto;
};

regresionLineal calculoRegresion(float *x, float *y)
{
    double m,b; //variables de pendiente e intercepto
    regresionLineal reg = regresionLineal();
    //suma de productos x,y
    double sumx =0;
    double sumy =0;
    double sumx_2 =0;
    double sumProd =0;
    for(int i =0; i<N; i++)
    {
        sumx += (double)x[i];
        sumy += (double)y[i];
        sumProd += (double)x[i]*y[i];
        sumx_2 += (double)x[i]*x[i];
    }
    /* calculo de la pendiente */
    m =(N*sumProd)-(sumx*sumy)/((N*sumx_2)-(sumx*sumx));
    /* calculo del intercepto */
    b = (sumy - (m*sumx))/N;
    reg.pendiente = m;
    reg.intercepto = b;
    return reg;
}

__global__ void porcentajeError(float *resultado, float *teorico, float *predic)//el array predic tiene 192 datos pero solo se usaran los primeros 96, el teorico tiene 96 datos
{
    int myID = threadIdx.x; //deberian ser 96?
    if(myID<T){
        resultado[myID]=abs(teorico[myID]-predic[myID])*100/teorico[myID];
    }
}

__global__ void prediccion(float *y, float *x, double m, double b)//y es el vector donde se guarda la prediccion
{
    y[(int)threadIdx.x] = ((float)m*x[(int)threadIdx.x]) + (float)b; //ecuacion de una recta
	printf("VALOR Y:%.2f\t VALOR X: %.2f \t M: %.2f\t B: %.2f\n", y[(int)threadIdx.x], x[(int)threadIdx.x], m, b);
	
}


int main(int argv, char* argc[])
{
    /* creacion streams, un stream por variable */
    hipStream_t stream1, stream2, stream3, stream4, stream5, stream6;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
	hipStreamCreate(&stream4);
    hipStreamCreate(&stream5);
    hipStreamCreate(&stream6);

    
    /* variables del host */
    float *dev_temp, *dev_hum, *dev_pres, *dev_secs, *dev_errorTemp, *dev_errorHum, *dev_errorPres; // pointers del device
    float *dev_temp3, *dev_hum3, *dev_pres3;
    float *temp, *hum, *pres, *secs, *temp_res, *hum_res, *pres_res, *temp3, *hum3, *pres3;
    string fechas[N], fechas3[T];
    //float temp3[T],hum3[T],pres3[T],fechas3[T];
    float errorTemp[T],errorHum[T],errorPres[T],secs3[T];
    
    /* reservas en memoria de los arrays a utilizar en host y pasar al device */
    hipHostAlloc( (void**)&fechas, N * sizeof(int), hipHostMallocDefault );// reserva de memoria de fechas
    hipHostAlloc( (void**)&secs, N * sizeof(int), hipHostMallocDefault );// reserva de memoria de segundos
    hipMalloc( (void**)&dev_secs, N * sizeof(int) );
	hipMalloc( (void**)&hum_res, N * sizeof(int) );
	hipMalloc( (void**)&temp_res, N * sizeof(int) );
	hipMalloc( (void**)&pres_res, N * sizeof(int) );


    
    //stream 1
    hipMalloc( (void**)&dev_temp, N * sizeof(int) );
	hipMalloc( (void**)&dev_errorTemp, N * sizeof(int) );
    hipHostAlloc( (void**)&temp, N * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&temp3, N * sizeof(int), hipHostMallocDefault );
	hipHostAlloc( (void**)&dev_temp3, N * sizeof(int), hipHostMallocDefault);

    
    //stream 2
    hipMalloc( (void**)&dev_hum, N * sizeof(int) );
	hipMalloc( (void**)&dev_errorHum, N * sizeof(int) );
	hipHostAlloc( (void**)&dev_hum3, N * sizeof(int), hipHostMallocDefault);
    hipHostAlloc( (void**)&hum, N * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&hum3, N * sizeof(int), hipHostMallocDefault);

    
    //stream 3
    hipMalloc( (void**)&dev_pres, N * sizeof(int) );
	hipMalloc( (void**)&dev_errorPres, N * sizeof(int) );
	hipHostAlloc( (void**)&dev_pres3, N * sizeof(int), hipHostMallocDefault);
    hipHostAlloc( (void**)&pres3, N * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&pres, N * sizeof(int), hipHostMallocDefault);
    

    /* lectura de datos del csv */
    int i = 0; //indice
    string humedad, presion, temperatura, altitud, fecha;
    ifstream file("datos.csv");

    while (getline(file, humedad, ',')) {
        //hay que revisar si no esta jalando los datos de altitud, los cuales no sirven
        std::size_t offset = 0;
        hum[i] = std::stod(humedad,&offset);
        offset = 0;
        getline(file, presion, ',') ;
        pres[i] = stod(presion,&offset);
        offset = 0;
        getline(file, temperatura, ',') ;
        temp[i] = std::stod(temperatura,&offset);
        getline(file, altitud, ',');
        getline(file, fecha);
        fechas[i] = fecha;
       // printf("\ni: %d, Humedad: %.2f, Presion: %.2f, Temp: %.2f, Fecha: %s", i, hum[i], pres[i], temp[i], fechas[i].c_str());
        i++;
    }

/* lectura de datos del csv del tercer dia*/

    i = 0; //indice
    ifstream file3("dia3.csv");
    string humedad3, presion3, temperatura3, altitud3, fecha3;
	printf("jfdkslajfkdslajfskd");
    while (getline(file3, humedad3, ',')) {
        //hay que revisar si no esta jalando los datos de altitud, los cuales no sirven
        std::size_t offset = 0;
        hum3[i] = std::stod(humedad3,&offset);
		//printf("humedad: %.2f", hum3[i]);
        offset = 0;
        getline(file3, presion3, ',') ;
        pres3[i] = stod(presion3,&offset);
		//printf("presion: %.2f", pres3[i]);
        offset = 0;
        getline(file3, temperatura3, ',') ;
        temp3[i] = stod(temperatura3,&offset);
		//printf("temperatura: %.2f", temp3[i]);
        getline(file3, altitud3, ',');
        getline(file3, fecha3);
        fechas3[i] = fecha3;
		//printf("fecha: %s\n", fechas3[i].c_str());
        printf("\ni: %d, Humedad: %.2f, Presion: %.2f, Temp: %.2f, Fecha: %s", i, hum3[i], pres3[i], temp3[i], fechas3[i].c_str());
        i++;
    }


    /* segundos de las primeras 48hr */
    for (int i =0; i <N; i++)
    {
        secs[i]=900*i;
	//Ya con los datos de los primeros dos dias
	  if(i<N/2){
        secs[i] = 900*i;}
	  else{
	   secs[i]=900*(i-N/2)+1;//ajuste para que el segundo dia no caiga en la misma hora
       }
    }


    /* calculo de la regresion lineal para temperatura, presion y humedad */
    regresionLineal regHum = calculoRegresion(hum, secs);
    regresionLineal regPres = calculoRegresion(pres, secs);
    regresionLineal regTemp = calculoRegresion(temp, secs);

    /* ajustar el vector de segundos para que ahora sean los segundos del tercer dia (prediccion) */
    for (int i =0; i <N; i++)
    {
        secs3[i] = (900*i); //48hr + segs del tercer dia
    }
    /* lanzamiento de kernels para la prediccion
        se lanzaran N threads en los que cada uno calculara la prediccion en la hora correspondiente de su variable correspondiente. */

    hipMemcpyAsync(dev_hum,hum,N*sizeof(int),hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(dev_pres,pres,N*sizeof(int),hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(dev_temp,temp,N*sizeof(int),hipMemcpyHostToDevice,stream3);
    hipMemcpyAsync(dev_secs,secs3,N*sizeof(int),hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(dev_secs,secs3,N*sizeof(int),hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(dev_secs,secs3,N*sizeof(int),hipMemcpyHostToDevice,stream3);
    prediccion<<<1, N, 0, stream1>>>(dev_hum, dev_secs, regHum.pendiente, regHum.intercepto);
    prediccion<<<1, N, 1, stream2>>>(dev_pres, dev_secs, regPres.pendiente, regPres.intercepto);
    prediccion<<<1, N, 2, stream3>>>(dev_temp, dev_secs, regTemp.pendiente, regTemp.intercepto);
	hipMemcpyAsync(hum_res,dev_hum,N*sizeof(int),hipMemcpyDeviceToHost,stream1);
    hipMemcpyAsync(pres_res,dev_pres,N*sizeof(int),hipMemcpyDeviceToHost,stream2);
    hipMemcpyAsync(temp_res,dev_temp,N*sizeof(int),hipMemcpyDeviceToHost,stream3);


/* realizacion y lanzamiento de kernels de porcentaje de error */
    hipMemcpyAsync(dev_hum,hum_res,N*sizeof(int),hipMemcpyHostToDevice,stream4);
    hipMemcpyAsync(dev_pres,pres_res,N*sizeof(int),hipMemcpyHostToDevice,stream5);
    hipMemcpyAsync(dev_temp,temp_res,N*sizeof(int),hipMemcpyHostToDevice,stream6);

	porcentajeError<<<1, T, 3, stream4>>>(dev_errorHum, dev_hum3, dev_hum);
	porcentajeError<<<1, T, 4, stream5>>>(dev_errorTemp, dev_temp3, dev_temp);
	porcentajeError<<<1, T, 5, stream6>>>(dev_errorPres, dev_pres3, dev_pres);

	hipMemcpyAsync(errorHum,dev_errorHum,N*sizeof(int),hipMemcpyDeviceToHost,stream4);
    hipMemcpyAsync(errorPres,dev_errorPres,N*sizeof(int),hipMemcpyDeviceToHost,stream5);
    hipMemcpyAsync(errorTemp,dev_errorTemp,N*sizeof(int),hipMemcpyDeviceToHost,stream6);
    /* display de prediccion o escritura en un nuevo .csv */
    //falta agregarle la fecha y hora para cada prediccion
    printf("\nPREDICCIONES\n");
    ofstream MiArchivo ("prediccion.csv");
    for(int i=0; i<T; i++)
    {
        if (MiArchivo.is_open())
        {
            MiArchivo <<hum_res[i]<<","<< errorHum[i]<<","<<pres_res[i]<<","<<errorPres[i]<<","<<temp_res[i]<<","<<errorTemp[i]<<","<<secs[i]<<"\n";
        }
		printf("H: %.2f (%.2f), P: %.2f (%.2f), T: %.2f (%.2f)\n", hum_res[i], errorHum[i], pres_res[i], errorPres[i], temp_res[i], errorTemp[i]);
    }
    MiArchivo.close();

    hipStreamSynchronize(stream1); // wait for stream1 to finish
    hipStreamSynchronize(stream2); // wait for stream2 to finish
    hipStreamSynchronize(stream3); // wait for stream2 to finish
    hipStreamSynchronize(stream5);
    hipStreamSynchronize(stream4);
    hipStreamSynchronize(stream6);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipStreamDestroy(stream4);
    hipStreamDestroy(stream5);
    hipStreamDestroy(stream6);


    hipFree(dev_hum);
    hipFree(dev_temp);
    hipFree(dev_pres);
    hipFree(dev_secs);

    hipFree(dev_errorTemp);
    hipFree(dev_errorPres);
    hipFree(dev_errorHum);
    return 0;
}
