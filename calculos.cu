/*

 Universidad del Valle de Guatemala
 CC3056
 Ana Lucia Hernandez. 17138.
 Andrea Arg�ello. 17801.
 Proyecto 3
 Programaci�n de Microprocesadores

*/


#include <iostream>
#include <stdio.h>
//#include <string>
//#include <string.h>
//#include <errno.h>
#include <stdint.h>
#include <stdlib.h>
#include <time.h>
//#include <io.h>
#include <hip/hip_runtime.h>
#include <fstream>

#define N 192 //cantidad de datos en los dos dias (serian 192)
#define T 96 //cantidad de datos del tercer dia (serian 96)
using namespace std;

struct regresionLineal
{
    double pendiente;
    double intercepto;
};

regresionLineal calculoRegresion(float *x, float *y)
{
    double m,b; //variables de pendiente e intercepto
    regresionLineal reg = regresionLineal();
    //suma de productos x,y
    double sumx =0;
    double sumy =0;
    double sumx_2 =0;
    double sumProd =0;
    for(int i =0; i<N; i++)
    {
        sumx += (double)x[i];
        sumy += (double)y[i];
        sumProd += (double)x[i]*y[i];
        sumx_2 += (double)x[i]*x[i];
    }
    sumx = sumx/2;
    /* calculo de la pendiente */
    m =((N*sumProd)-(sumx*sumy))/((N*sumx_2)-(sumx*sumx));
    /* calculo del intercepto */
    b = (sumy - (m*sumx))/N;
    reg.pendiente = m;
    reg.intercepto = b;
    return reg;
}

__global__ void porcentajeError(float *resultado, float *teorico, float *predic)//el array predic tiene 192 datos pero solo se usaran los primeros 96, el teorico tiene 96 datos
{
    printf("kernel % error");
    int myID = threadIdx.x; //deberian ser 96?
        resultado[myID]=abs(teorico[myID]-predic[myID])*100/teorico[myID];
        printf("\n\tTeorico: %.3f\tPrediccion: %.3f\tError: %.3f");
}

/*__global__ void prediccion(float *y, float *x)//y es el vector donde se guarda la prediccion
{
	float predic[T];
	int ia = threadIdx.x; //indice a (todos los valores del dia 1)
	int ib = (threadIdx.x) + (T); //indice b (todos los valores del dia 2)
	int xf = (threadIdx.x) + (2*T*900); //indice c (del valor y que queremos, en el dia 3)
	predic[ia] = y[ia] + ((xf-x[ia])*((y[ib]-y[ia])/(x[ib]- x[ia])));
    printf("VALOR Y:%.2f\t VALOR X: %.2f\tVALOR INDICE B: %.2f\tVALOR Y1: %.2f\tVALOR Y2: %.2f\n",predic[ia], x[ia], x[ib], y[ia],y[ib]);
	y[ia] = predic[ia];
	
}*/

__global__ void prediccion(float *y, float *x, float *x192, float *y192)//y es el vector donde se guarda la prediccion
{
    int myID= threadIdx.x;
    float m=(y192[myID]-y192[myID+T])/(x192[myID]-x192[myID+T]);
    float b=(m*x192[myID])+y192[myID];
    //regresionLineal myb = calculoRegresion(x192[myID],x192[myID+T],y192[myID],y192[myID+T]);
    //double m=myb.pendiente;
    //double b=myb.intercepto;
    y[myID] = ((float)m*x[myID]) + (float)b; //ecuacion de una recta

}


int main(int argv, char* argc[])
{
    /* creacion streams, un stream por variable */
    hipStream_t stream1, stream2, stream3, stream4, stream5, stream6;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
	hipStreamCreate(&stream4);
    hipStreamCreate(&stream5);
    hipStreamCreate(&stream6);


    /* variables del host */
    float *dev_temp, *dev_hum, *dev_pres, *dev_secs, *dev_secs3, *dev_errorTemp, *dev_errorHum, *dev_errorPres; // pointers del device
    float *dev_temp3, *dev_hum3, *dev_pres3, *dev_tempres, *dev_humres, *dev_presres;
    float *temp, *hum, *pres, *secs, *temp_res, *hum_res, *pres_res, *temp3, *hum3, *pres3;
  //  string *fechas, *fechas3;
    string fechas[N], fechas3[T];
    //float temp3[T],hum3[T],pres3[T],fechas3[T];
    float errorTemp[T],errorHum[T],errorPres[T],secs3[T];

    /* reservas en memoria de los arrays a utilizar en host y pasar al device */
  //  cudaHostAlloc( (void**)&fechas, N * sizeof(int), cudaHostAllocDefault );// reserva de memoria de fechas
   // cudaHostAlloc( (void**)&fechas3, N * sizeof(int), cudaHostAllocDefault );// reserva de memoria de fechas
    hipHostAlloc( (void**)&secs, N * sizeof(int), hipHostMallocDefault );// reserva de memoria de segundos
    hipMalloc( (void**)&dev_secs, N * sizeof(int) );
    hipMalloc( (void**)&dev_secs3, N * sizeof(int) );
    hipMalloc( (void**)&dev_humres, N * sizeof(int) );
    hipMalloc( (void**)&dev_tempres, N * sizeof(int) );
    hipMalloc( (void**)&dev_presres, N * sizeof(int) );

    //stream 1
    hipMalloc( (void**)&dev_temp, N * sizeof(int) );
    hipMalloc( (void**)&dev_temp3, N * sizeof(int) );
	hipMalloc( (void**)&dev_errorTemp, N * sizeof(int) );
    hipHostAlloc( (void**)&temp, N * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&temp_res, N * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&temp3, N * sizeof(int), hipHostMallocDefault );


    //stream 2
    hipMalloc( (void**)&dev_hum, N * sizeof(int) );
    hipMalloc( (void**)&dev_hum3, N * sizeof(int) );
	hipMalloc( (void**)&dev_errorHum, N * sizeof(int) );
    hipHostAlloc( (void**)&hum, N * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&hum_res, N * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&hum3, N * sizeof(int), hipHostMallocDefault);


    //stream 3
    hipMalloc( (void**)&dev_pres, N * sizeof(int) );
    hipMalloc( (void**)&dev_pres3, N * sizeof(int) );
	hipMalloc( (void**)&dev_errorPres, N * sizeof(int) );
    hipHostAlloc( (void**)&pres3, N * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&pres_res, N * sizeof(int), hipHostMallocDefault );
    hipHostAlloc( (void**)&pres, N * sizeof(int), hipHostMallocDefault);

    /* lectura de datos del csv */
    int i = 0; //indice
    string humedad, presion, temperatura, altitud, fecha;
    ifstream file("datos.csv");
    while (getline(file, humedad, ',')) {
        //hay que revisar si no esta jalando los datos de altitud, los cuales no sirven
        hum[i] = (float)atof(humedad.c_str());
        getline(file, presion, ',') ;
        pres[i] = (float)atof(presion.c_str());
        getline(file, temperatura, ',') ;
        temp[i] = (float)atof(temperatura.c_str());
        getline(file, altitud, ',');
        getline(file, fecha);
        fechas[i] = fecha;
        i++;
    }

/* lectura de datos del csv del tercer dia*/
    i = 0; //indice
    ifstream file3("dia3.csv");
    while (getline(file, humedad, ',')) {
        //hay que revisar si no esta jalando los datos de altitud, los cuales no sirven
        hum3[i] = (float)atof(humedad.c_str());
        getline(file, presion, ',') ;
        pres3[i] = (float)atof(presion.c_str());
        getline(file, temperatura, ',') ;
        temp3[i] = (float)atof(temperatura.c_str());
        getline(file, altitud, ',');
        getline(file, fecha);
        fechas3[i] = fecha;
        i++;
    }


    /* segundos de las primeras 48hr */
    for (int i =0; i <N; i++)
    {
        secs[i]=i;
    }

    /* calculo de la regresion lineal para temperatura, presion y humedad */
    regresionLineal regHum = calculoRegresion(hum, secs);
    regresionLineal regPres = calculoRegresion(pres, secs);
    regresionLineal regTemp = calculoRegresion(temp, secs);

    /* ajustar el vector de segundos para que ahora sean los segundos del tercer dia (prediccion) */
    for (int i =0; i <T; i++)
    {
        secs3[i] = (i+N); //48hr + segs del tercer dia
    }
    /* lanzamiento de kernels para la prediccion
        se lanzaran N threads en los que cada uno calculara la prediccion en la hora correspondiente de su variable correspondiente. */
    hipMemcpyAsync(dev_hum,hum,N*sizeof(int),hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(dev_pres,pres,N*sizeof(int),hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(dev_temp,temp,N*sizeof(int),hipMemcpyHostToDevice,stream3);
    hipMemcpyAsync(dev_secs,secs,N*sizeof(int),hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(dev_secs,secs,N*sizeof(int),hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(dev_secs,secs,N*sizeof(int),hipMemcpyHostToDevice,stream3);
    hipMemcpyAsync(dev_secs3,secs3,N*sizeof(int),hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(dev_secs3,secs3,N*sizeof(int),hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(dev_secs3,secs3,N*sizeof(int),hipMemcpyHostToDevice,stream3);

    hipStreamSynchronize(stream1); // wait for stream1 to finish
    hipStreamSynchronize(stream2); // wait for stream2 to finish
    hipStreamSynchronize(stream3); // wait for stream3 to finish

    hipMemcpyAsync(dev_hum3,hum3,N*sizeof(int),hipMemcpyHostToDevice,stream4);
    hipMemcpyAsync(dev_pres3,pres3,N*sizeof(int),hipMemcpyHostToDevice,stream5);
    hipMemcpyAsync(dev_temp3,temp3,N*sizeof(int),hipMemcpyHostToDevice,stream6);

    prediccion<<<1, T, 0, stream1>>>(dev_tempres, dev_secs3, dev_secs, dev_hum);
    prediccion<<<1, T, 1, stream2>>>(dev_presres, dev_secs3, dev_secs, dev_pres);
    prediccion<<<1, T, 2, stream3>>>(dev_tempres, dev_secs3, dev_secs, dev_temp);
	hipMemcpyAsync(hum_res,dev_humres,N*sizeof(int),hipMemcpyDeviceToHost,stream1);
    hipMemcpyAsync(pres_res,dev_presres,N*sizeof(int),hipMemcpyDeviceToHost,stream2);
    hipMemcpyAsync(temp_res,dev_tempres,N*sizeof(int),hipMemcpyDeviceToHost,stream3);

/* realizacion y lanzamiento de kernels de porcentaje de error */
    hipMemcpyAsync(dev_hum,hum_res,N*sizeof(int),hipMemcpyHostToDevice,stream4);
    hipMemcpyAsync(dev_pres,pres_res,N*sizeof(int),hipMemcpyHostToDevice,stream5);
    hipMemcpyAsync(dev_temp,temp_res,N*sizeof(int),hipMemcpyHostToDevice,stream6);
    hipMemcpyAsync(dev_hum3,hum3,N*sizeof(int),hipMemcpyHostToDevice,stream4);
    hipMemcpyAsync(dev_pres3,pres3,N*sizeof(int),hipMemcpyHostToDevice,stream5);
    hipMemcpyAsync(dev_temp3,temp3,N*sizeof(int),hipMemcpyHostToDevice,stream6);

	porcentajeError<<<1, T, 0, stream4>>>(dev_errorHum, dev_hum3, dev_humres);
	porcentajeError<<<1, T, 1, stream5>>>(dev_errorTemp, dev_temp3, dev_tempres);
	porcentajeError<<<1, T, 2, stream6>>>(dev_errorPres, dev_pres3, dev_presres);

	hipMemcpyAsync(errorHum,dev_errorHum,N*sizeof(int),hipMemcpyDeviceToHost,stream4);
    hipMemcpyAsync(errorPres,dev_errorPres,N*sizeof(int),hipMemcpyDeviceToHost,stream5);
    hipMemcpyAsync(errorTemp,dev_errorTemp,N*sizeof(int),hipMemcpyDeviceToHost,stream6);
    /* display de prediccion o escritura en un nuevo .csv */
    //falta agregarle la fecha y hora para cada prediccion
  /*  printf("\nPREDICCIONES\n");
    ofstream MiArchivo ("prediccion.csv");
    for(int i=0; i<T; i++)
    {
        if (MiArchivo.is_open())
        {
            printf("Guardando...\t");
            MiArchivo <<hum_res[i]<<","<< errorHum[i]<<","<<pres_res[i]<<","<<errorPres[i]<<","<<temp_res[i]<<","<<errorTemp[i]<<","<<secs[i]<<"\n";
            printf("H: %.2f (%.2f), P: %.2f (%.2f), T: %.2f (%.2f)\n", hum_res[i], errorHum[i], pres_res[i], errorPres[i], temp_res[i], errorTemp[i]);
        }
    }
    MiArchivo.close();*/
        /* display de prediccion o escritura en un nuevo .csv */
    //falta agregarle la fecha y hora para cada prediccion
    printf("\nTEMPERATURAS PREDICCION");
    for(int i=0; i<T; i++){
      printf("\nTiempo %d: %.3f",i,temp_res[i]);
    }
    printf("\nHUMEDADES PREDICCION");
    for(int i=0; i<T; i++){
      printf("\nTiempo %d: %.3f",i,hum_res[i]);
    }
    printf("\nPRESIONES PREDICCION");
    for(int i=0; i<T; i++){
      printf("\nTiempo %d: %.3f",i,pres_res[i]);
    }
    printf("\nPREDICCIONES\n");
    ofstream MiArchivo ("prediccion.csv");
    for(int i=0; i<T; i++)
    {
        if (MiArchivo.is_open())
        {
            printf("Guardando...");
            MiArchivo <<hum_res[i]<<","<< errorHum[i]<<","<<pres_res[i]<<","<<errorPres[i]<<","<<temp_res[i]<<","<<errorTemp[i]<<","<<secs[i]<<"\n";
            printf("H: %.2f (%.6f), P: %.2f (%.6f), T: %.2f (%.6f)\n", hum_res[i], errorHum[i], pres_res[i], errorPres[i], temp_res[i], errorTemp[i]);
        }
    }
    MiArchivo.close();

    hipStreamSynchronize(stream5);
    hipStreamSynchronize(stream4);
    hipStreamSynchronize(stream6);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipStreamDestroy(stream4);
    hipStreamDestroy(stream5);
    hipStreamDestroy(stream6);

    hipFree(dev_hum);
    hipFree(dev_temp);
    hipFree(dev_pres);
    hipFree(dev_secs);
    hipFree(dev_humres);
    hipFree(dev_tempres);
    hipFree(dev_presres);
    hipFree(dev_secs3);
    hipFree(dev_errorTemp);
    hipFree(dev_errorPres);
    hipFree(dev_errorHum);
    return 0;
}
